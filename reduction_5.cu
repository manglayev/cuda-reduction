#include "hip/hip_runtime.h"
#include "reduction_header.cuh"
#include ""
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

__device__ void warpReduce(volatile int* sdata, int tid);
__device__ int* reduction_5(int *g_idata, int *g_odata)
{
    static __shared__ int sdata[THREADS];
    // each thread loads one element from global to shared mem    
    unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;
    sdata[threadIdx.x] = g_idata[i] + g_idata[i+blockDim.x];
    __syncthreads();
    // do reduction in shared mem
    for(unsigned int s = blockDim.x/2; s>32; s>>=1)
    {
        if(threadIdx.x < s)
        {
          sdata[threadIdx.x] += sdata[threadIdx.x + s];
        }
        __syncthreads();
    }
    if (threadIdx.x < 32) warpReduce(sdata, threadIdx.x);
    // write result for this block to global mem
    if (threadIdx.x == 0) 
    {
        g_odata[blockIdx.x] = sdata[0];
    }
    //implement second reduction for the summed array
    if (threadIdx.x < 32) warpReduce(g_odata, threadIdx.x);
    if (threadIdx.x == 0) 
    {
        g_odata[blockIdx.x] = g_odata[0];
    }
    return g_odata;
}

__device__ void warpReduce(volatile int* sdata, int tid)
{
    sdata[tid] += sdata[tid + 32];
    sdata[tid] += sdata[tid + 16];
    sdata[tid] += sdata[tid + 8];
    sdata[tid] += sdata[tid + 4];
    sdata[tid] += sdata[tid + 2];
    sdata[tid] += sdata[tid + 1];
}