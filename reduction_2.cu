#include "hip/hip_runtime.h"
#include "reduction_header.cuh"
#include ""
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

__device__ int* reduction_2(int *g_idata, int *g_odata)
{
  __shared__ int sdata[THREADS];
  // each thread loads one element from global to shared mem
  unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
  sdata[threadIdx.x] = g_idata[i];
  __syncthreads();
  // do reduction in shared mem
  for(unsigned int s = 1; s < blockDim.x; s *= 2)
  {
    int index = 2 * s * threadIdx.x;
    if (index < blockDim.x)
    {
      sdata[index] += sdata[index + s];
    }
    __syncthreads();
  }
  // write result for this block to global mem
  if (threadIdx.x == 0)
  {
    g_odata[blockIdx.x] = sdata[0];
  }
  //implement second reduction for the summed array
  for(unsigned int s = 1; s < blockDim.x; s *= 2)
  {
    int index = 2 * s * threadIdx.x;
    if (index < blockDim.x)
    {
      g_odata[index] += g_odata[index + s];
    }
    __syncthreads();
  }
 // write result for this block to global mem
  if (threadIdx.x == 0) 
  {
    g_odata[blockIdx.x] = g_odata[0];
  }
  return g_odata;
}