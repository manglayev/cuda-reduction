#include "hip/hip_runtime.h"
#include "reduction_header.cuh"
#include ""
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

__device__ int* reduction_2(int *g_idata, int *g_odata);
__device__ int* reduction_2(int *g_idata, int *g_odata)
{
  extern __shared__ int sdata[];
  // each thread loads one element from global to shared mem
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
  sdata[tid] = g_idata[i];
  __syncthreads();
  // do reduction in shared mem
  for(unsigned int s=1; s < blockDim.x; s *= 2)
  {
    int index = 2 * s * tid;
    if (index < blockDim.x)
    {
      sdata[index] += sdata[index + s];
    }
    __syncthreads();
  }
  // write result for this block to global mem
  if (tid == 0) 
  {
    g_odata[blockIdx.x] = sdata[0];
  }
  return g_odata;
}