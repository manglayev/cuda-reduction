#include "hip/hip_runtime.h"
#include "reduction_header.cuh"
#include ""
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"


template<unsigned int blockSize>
__device__ int* reduction_6(int *g_idata, int *g_odata)
{
    static __shared__ int sdata[THREADS];
    // each thread loads one element from global to shared mem
    //unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;
    sdata[threadIdx.x] = g_idata[i] + g_idata[i+blockDim.x];
    __syncthreads();
    // do reduction in shared mem
    if (blockSize >= 512)
    {
        if (threadIdx.x < 256)
        {
            sdata[threadIdx.x] += sdata[threadIdx.x + 256];
        }
        __syncthreads();
    }
    if (blockSize >= 256)
    {
        if (threadIdx.x < 128)
        {
            sdata[threadIdx.x] += sdata[threadIdx.x + 128];
        }
        __syncthreads();
    }
    if (blockSize >= 128)
    {
        if (threadIdx.x < 64)
        {
            sdata[threadIdx.x] += sdata[threadIdx.x + 64];
        }
        __syncthreads();
    }
    if (threadIdx.x < 32) warpReduce<blockSize>(sdata, threadIdx.x);
    // write result for this block to global mem
    if (threadIdx.x == 0)
    {
        g_odata[blockIdx.x] = sdata[0];
    }
    __syncthreads();
    //implement second reduction for the summed array
    i = blockIdx.x*(blockDim.x*2) + threadIdx.x;
    sdata[threadIdx.x] = g_odata[i] + g_odata[i+blockDim.x];
    __syncthreads();
    if (blockSize >= 512)
    {
        if (threadIdx.x < 256)
        {
            sdata[threadIdx.x] += sdata[threadIdx.x + 256];
        }
        __syncthreads();
    }
    if (blockSize >= 256)
    {
        if (threadIdx.x < 128)
        {
            sdata[threadIdx.x] += sdata[threadIdx.x + 128];
        }
        __syncthreads();
    }
    if (blockSize >= 128)
    {
        if (threadIdx.x < 64)
        {
            sdata[threadIdx.x] += sdata[threadIdx.x + 64];
        }
        __syncthreads();
    }
    if (threadIdx.x < 32) warpReduce<blockSize>(sdata, threadIdx.x);
    __syncthreads();
    // write result for this block to global mem
    if (threadIdx.x == 0)
    {
        g_odata[blockIdx.x] = sdata[0];
    }
    return g_odata;
}

template<unsigned int blockSize>
__device__ int* reduction_61(int *g_idata, int *g_odata)
{
  static __shared__ int sdata[THREADS];
  // each thread loads one element from global to shared mem
  //unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;
  sdata[threadIdx.x] = g_idata[i] + g_idata[i+blockDim.x];
  __syncthreads();
  // do reduction in shared memory
  if (blockSize >= 1024)
  {
      if (threadIdx.x < 512)
      {
          sdata[threadIdx.x] += sdata[threadIdx.x + 512];
      }
      __syncthreads();
  }
  if (blockSize >= 512)
  {
      if (threadIdx.x < 256)
      {
          sdata[threadIdx.x] += sdata[threadIdx.x + 256];
      }
      __syncthreads();
  }
  if (blockSize >= 256)
  {
      if (threadIdx.x < 128)
      {
          sdata[threadIdx.x] += sdata[threadIdx.x + 128];
      }
      __syncthreads();
  }
  if (blockSize >= 128)
  {
      if (threadIdx.x < 64)
      {
          sdata[threadIdx.x] += sdata[threadIdx.x + 64];
      }
      __syncthreads();
  }
  if (threadIdx.x < 32) warpReduce<blockSize>(sdata, threadIdx.x);
  // write result for this block to global mem
  if (threadIdx.x == 0)
  {
      g_odata[blockIdx.x] = sdata[0];
  }
  return g_odata;
}

template<unsigned int blockSize>
__device__ int* reduction_62(int *g_idata, int *g_odata)
{
  static __shared__ int sdata[BLOCKS/4];
  // each thread loads one element from global to shared mem
  //unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;
  sdata[threadIdx.x] = g_idata[i] + g_idata[i+blockDim.x];
  __syncthreads();
  // do reduction in shared mem
  if (blockSize >= 512)
  {
      if (threadIdx.x < 256)
      {
          sdata[threadIdx.x] += sdata[threadIdx.x + 256];
      }
      __syncthreads();
  }
  if (blockSize >= 256)
  {
      if (threadIdx.x < 128)
      {
          sdata[threadIdx.x] += sdata[threadIdx.x + 128];
      }
      __syncthreads();
  }
  if (blockSize >= 128)
  {
      if (threadIdx.x < 64)
      {
          sdata[threadIdx.x] += sdata[threadIdx.x + 64];
      }
      __syncthreads();
  }
  if (threadIdx.x < 32) warpReduce<blockSize>(sdata, threadIdx.x);
  // write result for this block to global mem
  if (threadIdx.x == 0)
  {
      g_odata[blockIdx.x] = sdata[0];
  }
  __syncthreads();
  return g_odata;
}
