#include "hip/hip_runtime.h"
#include "reduction_header.cuh"
#include ""
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

__device__ int* reduction_30(int *g_idata, int *g_odata)
{
  __shared__ int sdata[THREADS];
  // each thread loads one element from global to shared mem
  unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
  sdata[threadIdx.x] = g_idata[i];
  __syncthreads();
  // do reduction in shared mem
  for(unsigned int s = blockDim.x/2; s > 0; s>>=1)
  {
    if (threadIdx.x < s)
    {
        sdata[threadIdx.x] += sdata[threadIdx.x + s];
    }
    __syncthreads();
  }
  // write result for this block to global mem
  if (threadIdx.x == 0)
  {
    g_odata[blockIdx.x] = sdata[0];
  }
  __syncthreads();
  return g_odata;
}
