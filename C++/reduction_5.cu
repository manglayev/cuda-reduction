#include "hip/hip_runtime.h"
#include "reduction_header.cuh"
#include ""
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

__device__ void warpReduce(volatile int* sdata, int tid);
__device__ int* reduction_5(int *g_idata, int *g_odata)
{
    static __shared__ int sdata[THREADS];
    // each thread loads one element from global to shared mem
    unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;
    sdata[threadIdx.x] = g_idata[i] + g_idata[i+blockDim.x];
    __syncthreads();
    // do reduction in shared mem
    for(unsigned int s = blockDim.x/2; s>32; s>>=1)
    {
        if(threadIdx.x < s)
        {
          sdata[threadIdx.x] += sdata[threadIdx.x + s];
        }
        __syncthreads();
    }
    if (threadIdx.x < 32) warpReduce(sdata, threadIdx.x);
    // write result for this block to global mem
    if (threadIdx.x == 0)
    {
        g_odata[blockIdx.x] = sdata[0];
    }
    //implement second reduction for the summed array
    __syncthreads();
    i = blockIdx.x*(blockDim.x*2) + threadIdx.x;
    sdata[threadIdx.x] = g_odata[i] + g_odata[i+blockDim.x];
    __syncthreads();
    for(unsigned int s = blockDim.x/2; s>32; s>>=1)
    {
        if(threadIdx.x < s)
        {
          sdata[threadIdx.x] += sdata[threadIdx.x + s];
        }
        __syncthreads();
    }
    if (threadIdx.x < 32) warpReduce(sdata, threadIdx.x);
    __syncthreads();
    if (threadIdx.x == 0)
    {
        g_odata[blockIdx.x] = sdata[0];
    }
    return g_odata;
}

__device__ int* reduction_51(int *g_idata, int *g_odata)
{
    static __shared__ int sdata[THREADS];
    // each thread loads one element from global to shared mem
    unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;
    sdata[threadIdx.x] = g_idata[i] + g_idata[i+blockDim.x];
    __syncthreads();
    // do reduction in shared mem
    for(unsigned int s = blockDim.x/2; s>32; s>>=1)
    {
        if(threadIdx.x < s)
        {
          sdata[threadIdx.x] += sdata[threadIdx.x + s];
        }
        __syncthreads();
    }
    if (threadIdx.x < 32) warpReduce(sdata, threadIdx.x);
    // write result for this block to global mem
    if (threadIdx.x == 0)
    {
        g_odata[blockIdx.x] = sdata[0];
    }
    return g_odata;
}

__device__ int* reduction_52(int *g_idata, int *g_odata)
{
    __shared__ int sdata[BLOCKS/4];
    // each thread loads one element from global to shared memory
    int i = blockIdx.x * (blockDim.x*2) + threadIdx.x;
    sdata[threadIdx.x] = g_idata[i] + g_idata[i+blockDim.x];
    __syncthreads();
    // do reduction in shared memory
    for(unsigned int s = BLOCKS/8; s>32; s>>=1)
    {
        if (threadIdx.x < s)
        {
          sdata[threadIdx.x] += sdata[threadIdx.x + s];
        }
        __syncthreads();
    }
    if (threadIdx.x < 32) warpReduce(sdata, threadIdx.x);
    // write result for this block to global memory
    if (threadIdx.x == 0)
    {
        g_odata[blockIdx.x] = sdata[0];
    }
    __syncthreads();
    return g_odata;
}

__device__ void warpReduce(volatile int* sdata, int tid)
{
    sdata[tid] += sdata[tid + 32];
    sdata[tid] += sdata[tid + 16];
    sdata[tid] += sdata[tid + 8];
    sdata[tid] += sdata[tid + 4];
    sdata[tid] += sdata[tid + 2];
    sdata[tid] += sdata[tid + 1];
}
