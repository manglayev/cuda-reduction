#include "hip/hip_runtime.h"
#include "reduction_header.cuh"
#include ""
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#include "reduction_warp.cu"
#include "reduction_1.cu"
#include "reduction_2.cu"
#include "reduction_3.cu"
#include "reduction_4.cu"
#include "reduction_5.cu"
#include "reduction_6.cu"
#include "reduction_7.cu"

__global__ void cuda_global(int *dev_a, int *dev_b)
{
  switch (VARIANT)
  {
    case 1:
      dev_b = reduction_1(dev_a, dev_b);
      break;
    case 2:
      dev_b = reduction_2(dev_a, dev_b);
      break;
    case 3:
      dev_b = reduction_3(dev_a, dev_b);
      break;
    case 4:
      dev_b = reduction_4(dev_a, dev_b);
      break;
    case 5:
      dev_b = reduction_5(dev_a, dev_b);
      break;
    case 6:
      dev_b = reduction_6<THREADS>(dev_a, dev_b);
      break;
    case 7:
      dev_b = reduction_7<THREADS>(dev_a, dev_b);      
      break;
    default:
      dev_b = reduction_1(dev_a, dev_b);
      break;
  }  
}

int* initArray()
{
  static int array[CUDASIZE];
  for(int i = 0; i < CUDASIZE; i++)
  {
    array[i] = 1;
  }
  return array;
}

int checkResults(int *a)
{
  int sum = 0;
  for(int i = 0; i < CUDASIZE; i++)
  {
    sum = sum + a[i];
  }
  return sum;
}

void wrapper()
{
  printf("STAGE 3 WRAPPER START\n");

  int *a = initArray();
  int b[1];

  int *dev_a;
  int *dev_b;

  hipMalloc((void**)&dev_a, CUDASIZE*sizeof(int));
  hipMalloc((void**)&dev_b, sizeof(int));

  hipMemcpy(dev_a, a, CUDASIZE*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b, sizeof(int), hipMemcpyHostToDevice);

  cuda_global<<<BLOCKS, THREADS>>>(dev_a, dev_b);
  hipMemcpy(b, dev_b, sizeof(int), hipMemcpyDeviceToHost);
  printf("GPU RESULTS: b = %d\n", b[0]);
  int sum = checkResults(a);
  printf("CPU RESULTS: sum = %d\n", sum);

  hipFree(dev_a);
  hipFree(dev_b);
  printf("STAGE 3 WRAPPER END\n");
}
