#include "hip/hip_runtime.h"
#include "reduction_header.cuh"
#include ""
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#include "reduction_warp.cu"
#include "reduction_1.cu"
#include "reduction_2.cu"
#include "reduction_3.cu"
#include "reduction_4.cu"
//#include "reduction_5.cu"
#include "reduction_6.cu"
#include "reduction_7.cu"

#include "reduction_10.cu"
#include "reduction_20.cu"
#include "reduction_30.cu"
#include "reduction_41.cu"
#include "reduction_42.cu"
#include "reduction_50.cu"

__global__ void cuda_global(int *dev_a, int *dev_b)
{
  switch (VARIANT)
  {
    case 1:
      //dev_b = reduction_1(dev_a, dev_b);
      dev_b = reduction_10(dev_a, dev_b);
      break;
    case 2:
      //dev_b = reduction_2(dev_a, dev_b);
      dev_b = reduction_20(dev_a, dev_b);
      break;
    case 3:
      //dev_b = reduction_3(dev_a, dev_b);
      dev_b = reduction_30(dev_a, dev_b);
      break;
    case 4:
      //dev_b = reduction_4(dev_a, dev_b);
      if(blockDim.x == THREADS)
        dev_b = reduction_41(dev_a, dev_b);
      if(blockDim.x == BLOCKS/4)
        dev_b = reduction_42(dev_a, dev_b);
      break;
    case 5:
      //dev_b = reduction_5(dev_a, dev_b);
      if(blockDim.x == THREADS)
        dev_b = reduction_51(dev_a, dev_b);
      if(blockDim.x == BLOCKS/4)
        dev_b = reduction_52(dev_a, dev_b);
      break;
    case 6:
      dev_b = reduction_6<THREADS>(dev_a, dev_b);
      break;
    case 7:
      dev_b = reduction_7<THREADS>(dev_a, dev_b);
      break;
    default:
      dev_b = reduction_1(dev_a, dev_b);
      break;
  }
}

int* initArray()
{
  static int array[CUDASIZE];
  for(int i = 0; i < CUDASIZE; i++)
  {
    array[i] = 1;
  }
  return array;
}

int checkResults(int *a)
{
  int sum = 0;
  for(int i = 0; i < CUDASIZE; i++)
  {
    sum = sum + a[i];
  }
  return sum;
}

void wrapper()
{
  printf("STAGE 3 WRAPPER START\n");
  /*
  hipDeviceProp_t device;
  hipGetDeviceProperties(&device, 0);
  printf("  --- General information for device START ---\n");
  printf("Name: %s;\n", device.name);
  printf("Compute capability: %d.%d\n", device.major, device.minor);
  printf("Clock rate: %d\n", device.clockRate);
  printf("Total global memory: %ld\n", device.totalGlobalMem);
  printf("Total constant memory: %ld\n", device.totalConstMem);
  printf("Multiprocessor count: %d\n", device.multiProcessorCount);
  printf("Shared memory per block: %ld\n", device.sharedMemPerBlock);
  printf("Registers per block: %d\n", device.regsPerBlock);
  printf("Threads in warp: %d\n", device.warpSize);
  printf("Max threads Per Block: %d\n", device.maxThreadsPerBlock);
  printf("Max thread dimensions: (%d, %d, %d)\n", device.maxThreadsDim[0], device.maxThreadsDim[1], device.maxThreadsDim[2]);
  printf("Max grid dimensions: (%d, %d, %d)\n", device.maxGridSize[0], device.maxGridSize[1], device.maxGridSize[2]);
  printf("  --- General information for device END ---\n");
  */
  hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

  int *a = initArray();
  int b[1];
  int *dev_a;
  int *dev_b;

  hipMalloc((void**)&dev_a, CUDASIZE*sizeof(int));
  hipMemcpy(dev_a, a, CUDASIZE*sizeof(int), hipMemcpyHostToDevice);

  if(VARIANT < 4)
  {
    //hipMalloc((void**)&dev_b, sizeof(int));
    hipMalloc((void**)&dev_b, BLOCKS*sizeof(int));
    //hipMemcpy(dev_b, b, sizeof(int), hipMemcpyHostToDevice);
    //hipMemcpy(dev_b, b, BLOCKS*sizeof(int), hipMemcpyHostToDevice);
  }
  else
  {
    //hipMalloc((void**)&dev_b, sizeof(int));
    hipMalloc((void**)&dev_b, BLOCKS/2*sizeof(int));
    //hipMemcpy(dev_b, b, sizeof(int), hipMemcpyHostToDevice);
    //hipMemcpy(dev_b, b, BLOCKS/2*sizeof(int), hipMemcpyHostToDevice);
  }

  switch(VARIANT)
  {
    case 1:
    {
      cuda_global<<<BLOCKS, THREADS>>>(dev_a, dev_b);
      cuda_global<<<1, BLOCKS>>>(dev_b, dev_b);
      break;
    }
    case 2:
    {
      cuda_global<<<BLOCKS, THREADS>>>(dev_a, dev_b);
      cuda_global<<<1, BLOCKS>>>(dev_b, dev_b);
      break;
    }
    case 3:
    {
      cuda_global<<<BLOCKS, THREADS>>>(dev_a, dev_b);
      cuda_global<<<1, BLOCKS>>>(dev_b, dev_b);
      break;
    }
    case 4:
    {
      cuda_global<<<BLOCKS/2, THREADS>>>(dev_a, dev_b);
      cuda_global<<<1, BLOCKS/4>>>(dev_b, dev_b);
      break;
    }
    case 5:
    {
      cuda_global<<<BLOCKS/2, THREADS>>>(dev_a, dev_b);
      cuda_global<<<1, BLOCKS/4>>>(dev_b, dev_b);
      break;
    }
    default:
    {
      cuda_global<<<BLOCKS, THREADS>>>(dev_a, dev_b);
      cuda_global<<<1, BLOCKS>>>(dev_b, dev_b);
      break;
    }
  }

  //hipMemcpy(b, dev_b, sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(b, dev_b, sizeof(int), hipMemcpyDeviceToHost);
  hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
  printf("GPU RESULTS: VARIANT = %d; elapsed time: %.5f ms; \n", VARIANT, elapsedTime);
  printf("GPU RESULTS: sum = %d \n", b[0]);
  int sum = checkResults(a);
  printf("CPU RESULTS: sum = %d\n", sum);
  hipFree(dev_a);
  hipFree(dev_b);
  printf("STAGE 3 WRAPPER END\n");
}
