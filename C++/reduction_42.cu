#include "hip/hip_runtime.h"
#include "reduction_header.cuh"
#include ""
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

__device__ int* reduction_42(int *g_idata, int *g_odata)
{
    __shared__ int sdata[BLOCKS/4];
    // each thread loads one element from global to shared memory
    int i = blockIdx.x * (blockDim.x*2) + threadIdx.x;
    sdata[threadIdx.x] = g_idata[i] + g_idata[i+blockDim.x];
    __syncthreads();
    // do reduction in shared memory
    for(unsigned int s = BLOCKS/8; s>0; s>>=1)
    {
        if (threadIdx.x < s)
        {
          sdata[threadIdx.x] += sdata[threadIdx.x + s];
        }
        __syncthreads();
    }
    // write result for this block to global memory
    if (threadIdx.x == 0)
    {
        g_odata[blockIdx.x] = sdata[0];
    }
    __syncthreads();
    return g_odata;
}
