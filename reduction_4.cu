#include "hip/hip_runtime.h"
#include "reduction_header.cuh"
#include ""
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

__device__ int* reduction_4(int *g_idata, int *g_odata);
__device__ int* reduction_4(int *g_idata, int *g_odata)
{
    extern __shared__ int sdata[];
    // each thread loads one element from global to shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;
    sdata[tid] = g_idata[i] + g_idata[i+blockDim.x];
    __syncthreads();
    // do reduction in shared mem
    for(unsigned int s=blockDim.x/2; s>0; s>>=1)
    {
        if (tid < s)
        {
          sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }
    // write result for this block to global mem
    if (tid == 0) 
    {
        g_odata[blockIdx.x] = sdata[0];
    }
    return g_odata;
}