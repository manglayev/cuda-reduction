#include "hip/hip_runtime.h"
#include "reduction_header.cuh"
#include ""
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

__device__ int* reduction_4(int *g_idata, int *g_odata)
{
    __shared__ int sdata[THREADS];
    // each thread loads one element from global to shared mem
    unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;
    sdata[threadIdx.x] = g_idata[i] + g_idata[i+blockDim.x];
    __syncthreads();
    // do reduction in shared mem
    for(unsigned int s = blockDim.x/2; s>0; s>>=1)
    {
        if (threadIdx.x < s)
        {
          sdata[threadIdx.x] += sdata[threadIdx.x + s];
        }
        __syncthreads();
    }
    // write result for this block to global mem
    if (threadIdx.x == 0) 
    {
        g_odata[blockIdx.x] = sdata[0];
    }
    //implement second reduction for the summed array
    for(unsigned int s = blockDim.x/2; s>0; s>>=1)
    {
        if (threadIdx.x < s)
        {
          g_odata[threadIdx.x] += g_odata[threadIdx.x + s];
        }
        __syncthreads();
    }
    if (threadIdx.x == 0) 
    {
        g_odata[blockIdx.x] = g_odata[0];
    }
    return g_odata;
}