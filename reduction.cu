#include "hip/hip_runtime.h"
#include "reduction_header.cuh"
#include ""
#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <utility>
#include <type_traits>
#include <stdio.h>
#include <stdlib.h>

#include "reduction_1.cu"

__global__ void cuda_global(int *dev_a, int *dev_b)
{    
  dev_b = reduction_1(dev_a, dev_b);
}

int checkResults(int *a)
{
  int sum = 0;
  for(int i = 0; i < CUDASIZE; i++)
  {
    sum = sum + a[i];
  }
  return sum;
}

void wrapper()
{
  printf("STAGE 3 WRAPPER START\n");

  int a[CUDASIZE] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10};
  int b[1];

  int *dev_a;
  int *dev_b;

  hipMalloc((void**)&dev_a, CUDASIZE*sizeof(int));
  hipMalloc((void**)&dev_b, sizeof(int));

  hipMemcpy(dev_a, a, CUDASIZE*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b, sizeof(int), hipMemcpyHostToDevice);

  cuda_global<<<BLOCKS, THREADS>>>(dev_a, dev_b);

  hipMemcpy(b, dev_b, sizeof(int), hipMemcpyDeviceToHost);
  printf("GPU RESULTS: b = %d\n", b[0]);
  int sum = checkResults(a);
  printf("CPU RESULTS: sum = %d\n", sum);

  hipFree(dev_a);
  hipFree(dev_b);
  printf("STAGE 3 WRAPPER END\n");
}
