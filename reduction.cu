#include "hip/hip_runtime.h"
#include "reduction_header.cuh"
#include ""
#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <utility>
#include <type_traits>
#include <stdio.h>
#include <stdlib.h>

__global__ void cuda_global(int *g_idata, int *g_odata)
{
  extern __shared__ int sdata[];
  // each thread loads one element from global to shared mem
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
  sdata[tid] = g_idata[i];
  __syncthreads();
  // do reduction in shared mem
  for(unsigned int s=1; s < blockDim.x; s *= 2) {
  if (tid % (2*s) == 0) {
  sdata[tid] += sdata[tid + s];
  }
  __syncthreads();
  }
  // write result for this block to global mem
  if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

int checkResults(int *a)
{
  int sum = 0;
  for(int i = 0; i < CUDASIZE; i++)
  {
    sum = sum + a[i];
  }
  return sum;
}

void wrapper()
{
  printf("STAGE 3 WRAPPER START\n");

  int a[CUDASIZE] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10};
  int b[1];

  int *dev_a;
  int *dev_b;

  hipMalloc((void**)&dev_a, CUDASIZE*sizeof(int));
  hipMalloc((void**)&dev_b, sizeof(int));

  hipMemcpy(dev_a, a, CUDASIZE*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b, sizeof(int), hipMemcpyHostToDevice);

  cuda_global<<<BLOCKS, THREADS>>>(dev_a, dev_b);

  hipMemcpy(b, dev_b, sizeof(int), hipMemcpyDeviceToHost);
  printf("GPU RESULTS: b = %d\n", b[0]);
  int sum = checkResults(a);
  printf("CPU RESULTS: sum = %d\n", sum);

  hipFree(dev_a);
  hipFree(dev_b);
  printf("STAGE 3 WRAPPER END\n");
}
