#include "hip/hip_runtime.h"
#include "reduction_header.cuh"
#include ""
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"


template<unsigned int blockSize>
__device__ int* reduction_6(int *g_idata, int *g_odata)
{
    static __shared__ int sdata[THREADS];
    // each thread loads one element from global to shared mem
    //unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;
    sdata[threadIdx.x] = g_idata[i] + g_idata[i+blockDim.x];
    __syncthreads();
    // do reduction in shared mem
    if (blockSize >= 512)
    {
        if (threadIdx.x < 256)
        {
            sdata[threadIdx.x] += sdata[threadIdx.x + 256];
        }
        __syncthreads();
    }
    if (blockSize >= 256)
    {
        if (threadIdx.x < 128)
        {
            sdata[threadIdx.x] += sdata[threadIdx.x + 128];
        }
        __syncthreads();
    }
    if (blockSize >= 128)
    {
        if (threadIdx.x < 64)
        {
            sdata[threadIdx.x] += sdata[threadIdx.x + 64];
        }
        __syncthreads();
    }
    if (threadIdx.x < 32)
        warpReduce<blockSize>(sdata, threadIdx.x);
    // write result for this block to global mem
    if (threadIdx.x == 0)
    {
        g_odata[blockIdx.x] = sdata[0];
    }
    //implement second reduction for the summed array
    if (threadIdx.x < 32)
        warpReduce<blockSize>(g_odata, threadIdx.x);
    // write result for this block to global mem
    if (threadIdx.x == 0) 
    {
        g_odata[blockIdx.x] = g_odata[0];
    }    
    return g_odata;
}